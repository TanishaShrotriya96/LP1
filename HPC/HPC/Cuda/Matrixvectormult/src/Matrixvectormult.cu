/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include<hip/hip_runtime.h>
#include <stdlib.h>
#include<time.h>

#define SIZE 3

__global__ void matrixvectmult(int *mat,int *vect,int *res)
{
	int tid=blockIdx.x*blockDim.x;
	int mult=0;
	for(int i=0;i<SIZE;i++)
	{
		mult=mult+(mat[tid+i]*vect[i]);
	}
	res[blockIdx.x]=mult;
}

int main(void)
{
	int i,j;
	srand(time(NULL));
	int a[SIZE][SIZE],b[SIZE],c[SIZE];

	int *dev_a,*dev_b,*dev_c;

	hipMalloc((void **)&dev_a, SIZE*SIZE*sizeof(int));
	hipMalloc((void **)&dev_b, SIZE*sizeof(int));
	hipMalloc((void **)&dev_c, SIZE*sizeof(int));

	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			a[i][j] = rand()%20+1;
		}
	}

	printf("\nThe matrix is:\n");
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			printf("%d\t",a[i][j]);
		}
		printf("\n");
	}

	for(i=0;i<SIZE;i++)
	{
		b[i] = rand()%20+1;
	}

	printf("\nThe vector is:\n");
	for(i=0;i<SIZE;i++)
	{
		printf("%d  ",b[i]);
	}

	hipMemcpy(dev_a,a,sizeof(a),hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,sizeof(b),hipMemcpyHostToDevice);
	matrixvectmult<<<SIZE,SIZE>>>(dev_a,dev_b,dev_c);
	hipMemcpy(&c,dev_c,sizeof(c),hipMemcpyDeviceToHost);

	printf("\nThe result is:\n");
	for(int i=0;i<SIZE;i++)
	{
		printf("%d ",c[i]);
	}


	return 0;
}
